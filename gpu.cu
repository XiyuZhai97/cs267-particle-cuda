#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;
int binPerRow;
int binCount;
double binSize;
static texture<int2, 1, hipReadModeElementType> old_pos_tex;
static texture<int2, 1, hipReadModeElementType> old_vel_tex;
static texture<int2, 1, hipReadModeElementType> old_acc_tex;
static texture<int,  1, hipReadModeElementType> bin_index_tex;
static texture<int,  1, hipReadModeElementType> particle_index_tex;
static texture<int,  1, hipReadModeElementType> bin_start_tex;
static texture<int,  1, hipReadModeElementType> bin_end_tex;
double *d_pos;
double *d_vel;
double *d_acc;
double *sorted_pos;
double *sorted_vel;
double *sorted_acc;
int *bin_index;
int *particle_index;
int *original_p_index;
int *bin_start;
int *bin_end;
int num_bins;

static __inline__ __device__ double fetch_double(texture<int2, 1> t, int i)
{
	int2 v = tex1Dfetch(t, i);
	return __hiloint2double(v.y, v.x);
}

__global__ void copyparts_2( particle_t* parts, int num_parts, double *pos, double *vel, double* acc)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_parts) return;

	particle_t* p = &parts[tid];

	pos[2*tid] = p -> x;
	pos[2*tid+1] = p -> y;
	vel[2*tid] = p -> vx;
	vel[2*tid+1] = p -> vy;
	acc[2*tid] = p -> ax;
	acc[2*tid+1] = p -> ay;
    
}
__global__ void copyparts_back( particle_t* parts, int num_parts, double *pos, double *vel, double* acc)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_parts) return;

	particle_t* p = &parts[tid];

	p -> x = pos[2*tid];
	p -> y = pos[2*tid+1];
	p -> vx = vel[2*tid];
	p -> vy = vel[2*tid+1];
	p -> ax = acc[2*tid];
	p -> ay = acc[2*tid+1];
}
void sort_particles(int *bin_index, int *particle_index, int num_parts)
{
	thrust::sort_by_key(thrust::device_ptr<int>(bin_index),
			thrust::device_ptr<int>(bin_index + num_parts),
			thrust::device_ptr<int>(particle_index));
}

static __inline__ __device__ int binNum(double &d_x, double &d_y, int binPerRow, double binSize) 
{
	int rowid = int(d_y / binSize);
	int colid = int(d_x / binSize);

	if (rowid >= binPerRow)
		rowid = binPerRow - 1;
	if (colid >= binPerRow)
		colid = binPerRow - 1;
	return rowid * binPerRow + colid;
}

__global__ void reorder_data_calc_bin(int *bin_start, int *bin_end, double *sorted_pos, 
	double *sorted_vel, double *sorted_acc, int *bin_index, int *particle_index, int *original_p_index, double *d_pos, 
	double *d_vel, double *d_acc, int num_parts, int num_bins)
{
	extern __shared__ int sharedHash[];    // blockSize + 1 elements
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int bi;
	if (index < num_parts) {
		bi = bin_index[index];
		sharedHash[threadIdx.x+1] = bi;
		if (index > 0 && threadIdx.x == 0)
		{
			// first thread in block must load neighbor particle hash
			sharedHash[0] = bin_index[index-1];
		}
	}

	__syncthreads();

	if (index < num_parts) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell,
		// so store the index of this particle in the cell.
		// As it isn't the first particle, it must also be the cell end of
		// the previous particle's cell

		if (index == 0 || bi != sharedHash[threadIdx.x])
		{
			bin_start[bi] = index;
			if (index > 0)
				bin_end[sharedHash[threadIdx.x]] = index;
		}

		if (index == num_parts - 1)
		{
			bin_end[bi] = index + 1;
		}

		// Now use the sorted index to reorder the pos and vel data
		int sortedIndex = particle_index[index];
		sorted_pos[2*index]   = d_pos[2*sortedIndex];
		sorted_pos[2*index+1] = d_pos[2*sortedIndex+1];
		sorted_vel[2*index]   = d_vel[2*sortedIndex];
		sorted_vel[2*index+1] = d_vel[2*sortedIndex+1];
		sorted_acc[2*index]   = d_acc[2*sortedIndex];
		sorted_acc[2*index+1] = d_acc[2*sortedIndex+1];
		original_p_index[sortedIndex] = index;

	}
}
__global__ void reorder_data_back(double *sorted_back_pos, double *sorted_back_vel, double *sorted_back_acc, 
	int *original_p_index, double *sorted_pos, double *sorted_vel, double *sorted_acc, int num_parts)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < num_parts) {
		// Now use the sorted index to reorder the pos and vel data
		int sortedIndex = original_p_index[index];
		sorted_back_pos[2*index]     = sorted_pos[2*sortedIndex];
		sorted_back_pos[2*index+1] = sorted_pos[2*sortedIndex+1];
		sorted_back_vel[2*index]       = sorted_vel[2*sortedIndex];
		sorted_back_vel[2*index+1]   = sorted_vel[2*sortedIndex+1];
		sorted_back_acc[2*index]   = sorted_acc[2*sortedIndex];
		sorted_back_acc[2*index+1] = sorted_acc[2*sortedIndex+1];
	}
}

__global__ void calculate_bin_index(int *bin_index, int *particle_index, double *d_pos, int num_parts, int binPerRow, double binSize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index >= num_parts) return;
	double pos_x = fetch_double(old_pos_tex, 2*index);
	double pos_y = fetch_double(old_pos_tex, 2*index+1);
	int cbin = binNum( pos_x,pos_y,binPerRow, binSize);
	bin_index[index] = cbin;
	particle_index[index] = index;
}

__device__ void apply_force_gpu(double &particle_x, double &particle_y, double &particle_ax, double &particle_ay, double &neighbor_x, double &neighbor_y)
{
	double dx = neighbor_x - particle_x;
	double dy = neighbor_y - particle_y;
	double r2 = dx * dx + dy * dy;
	if( r2 > cutoff * cutoff )
		return;

	r2 = (r2 > min_r * min_r) ? r2 : min_r*min_r;
	double r = sqrt( r2 );

	//
	//  very simple short-range repulsive force
	//
	double coef = ( 1 - cutoff / r ) / r2 / mass;
	particle_ax += coef * dx;
	particle_ay += coef * dy;
}

__global__ void compute_forces_gpu(double *pos, double *acc, int num_parts, int binPerRow, int *bin_start, int *bin_end, double binSize)
{
	// Get thread (particle) ID
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= num_parts) 
		return;

	double pos_1x = fetch_double(old_pos_tex, 2*tid);
	double pos_1y = fetch_double(old_pos_tex, 2*tid+1);

	// find current particle's in, handle boundaries
	int cbin = binNum( pos_1x, pos_1y, binPerRow, binSize);
	// printf("Thread %d, cbin = %d, binPerRow = %d\n", tid, cbin, binPerRow);

	int lowi = -1, highi = 1, lowj = -1, highj = 1;
	if (cbin < binPerRow) // in the first row
		lowj = 0;
	if (cbin % binPerRow == 0) // in the first column
		lowi = 0;
	if (cbin % binPerRow == (binPerRow-1))
		highi = 0;
	if (cbin >= binPerRow*(binPerRow-1))
		highj = 0;

	double acc_x;
	double acc_y;
	acc_x = acc_y = 0;
	for (int i = lowi; i <= highi; i++)
		for (int j = lowj; j <= highj; j++)
		{
			int nbin = cbin + i + binPerRow*j;
			int bin_st = tex1Dfetch(bin_start_tex, nbin);
			if (bin_st != 0xffffffff) {
				int bin_et = tex1Dfetch(bin_end_tex, nbin);
				for (int k = bin_st; k < bin_et; k++ ) {
					double pos_2x = fetch_double(old_pos_tex, 2*k);
					double pos_2y = fetch_double(old_pos_tex, 2*k+1);
					// printf("Thread %d, neighbor_bin = %d\n", tid, k);

					apply_force_gpu( pos_1x, pos_1y, acc_x, acc_y, pos_2x, pos_2y );
				}
			}
		}
	acc[2*tid] = acc_x;
	acc[2*tid+1] = acc_y;
	// printf("Thread %d, acc_x = %f, acc_y = %f\n", tid, acc_x, acc_y);

}

__global__ void move_gpu (double *pos, double *vel, double *acc, int num_parts, double size)
{

	// Get thread (particle) ID
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_parts) 
		return;

	double acc_x = fetch_double(old_acc_tex, 2*tid);
	double acc_y = fetch_double(old_acc_tex, 2*tid+1);
	double vel_x = fetch_double(old_vel_tex, 2*tid);
	double vel_y = fetch_double(old_vel_tex, 2*tid+1);
	double pos_x = fetch_double(old_pos_tex, 2*tid);
	double pos_y = fetch_double(old_pos_tex, 2*tid+1);
	vel_x += acc_x * dt;
	vel_y += acc_y * dt;
	pos_x += vel_x * dt;
	pos_y += vel_y * dt;

	//
	//  bounce from walls
	//
	while( pos_x < 0 || pos_x > size )
	{
		pos_x = pos_x < 0 ? - (pos_x) : 2 * size - pos_x;
		vel_x = - (vel_x);
	}
	while( pos_y < 0 || pos_y > size )
	{
		pos_y = pos_y < 0 ? - (pos_y) : 2 * size - pos_y;
		vel_y = - (vel_y);
	}

	vel[2*tid] = vel_x;
	vel[2*tid+1] = vel_y;
	pos[2*tid] = pos_x;
    pos[2*tid+1] = pos_y;
	acc[2*tid] = 0;
	acc[2*tid+1] = 0;
}


void init_simulation(particle_t* parts, int num_parts, double size) 
{
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

	// GPU particle data structure
	hipMalloc((void **) &d_pos, 2*num_parts * sizeof(double));
	hipMalloc((void **) &d_vel, 2*num_parts * sizeof(double));
	hipMalloc((void **) &d_acc, 2*num_parts * sizeof(double));

	hipMalloc((void **) &sorted_pos, 2*num_parts * sizeof(double));
	hipMalloc((void **) &sorted_vel, 2*num_parts * sizeof(double));
	hipMalloc((void **) &sorted_acc, 2*num_parts * sizeof(double));
	
	hipMalloc((void **) &bin_index, num_parts * sizeof(int));
	hipMemset(bin_index, 0x0, num_parts * sizeof(int));
	hipMalloc((void **) &particle_index, num_parts * sizeof(int));
	hipMemset(particle_index, 0x0, num_parts * sizeof(int));

	hipMalloc((void **) &original_p_index, num_parts * sizeof(int));
	hipMemset(original_p_index, 0x0, num_parts * sizeof(int));

	binPerRow = int(size / (1.3 * cutoff));
	num_bins = binPerRow * binPerRow;

	hipMalloc((void **) &bin_start, num_bins * sizeof(int));
	hipMalloc((void **) &bin_end, num_bins * sizeof(int));
	hipMemset(bin_start, 0x0, num_bins * sizeof(int));
	hipMemset(bin_end, 0x0, num_bins * sizeof(int));
	hipDeviceSynchronize();
	blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
}

void simulate_one_step(particle_t* parts, int num_parts, double size) 
{
	copyparts_2 <<< blks, NUM_THREADS >>> (parts, num_parts, d_pos, d_vel, d_acc);

    hipBindTexture(0, old_pos_tex, d_pos, 2*num_parts * sizeof(int2));
    calculate_bin_index <<< blks, NUM_THREADS >>> (bin_index, particle_index, d_pos, num_parts, binPerRow, binSize);
    hipUnbindTexture(old_pos_tex);

    hipBindTexture(0, bin_index_tex, bin_index, num_parts * sizeof(int));
    hipBindTexture(0, particle_index_tex, particle_index, num_parts * sizeof(int));
	sort_particles(bin_index, particle_index, num_parts);
    hipUnbindTexture(bin_index_tex);
    hipUnbindTexture(particle_index_tex);

    hipMemset(bin_start, 0xffffffff, num_bins * sizeof(int));
    int smemSize = sizeof(int)*(NUM_THREADS+1);
    reorder_data_calc_bin <<< blks, NUM_THREADS, smemSize >>> (bin_start, bin_end, sorted_pos, sorted_vel, sorted_acc, bin_index, particle_index, original_p_index, d_pos, d_vel, d_acc, num_parts, num_bins);

    hipBindTexture(0, old_pos_tex, sorted_pos, 2*num_parts * sizeof(int2));
    hipBindTexture(0, bin_start_tex, bin_start, num_bins * sizeof(int));
    hipBindTexture(0, bin_end_tex, bin_end, num_bins * sizeof(int));

    compute_forces_gpu <<< blks, NUM_THREADS >>> (sorted_pos, sorted_acc, num_parts, binPerRow, bin_start, bin_end, binSize);

    hipUnbindTexture(old_pos_tex);
    hipUnbindTexture(bin_start_tex);
    hipUnbindTexture(bin_end_tex);


    hipDeviceSynchronize(); //or add hasMoved to parts

    // Move particles
    hipBindTexture(0, old_pos_tex, sorted_pos, 2*num_parts * sizeof(int2));
    hipBindTexture(0, old_vel_tex, sorted_vel, 2*num_parts * sizeof(int2));
    hipBindTexture(0, old_acc_tex, sorted_acc, 2*num_parts * sizeof(int2));
	move_gpu <<< blks, NUM_THREADS >>> (sorted_pos, sorted_vel, sorted_acc, num_parts, size);
    hipUnbindTexture(old_pos_tex);
    hipUnbindTexture(old_vel_tex);
    hipUnbindTexture(old_acc_tex);

    // Swap particles between d_particles and sorted_particles
	reorder_data_back<<< blks, NUM_THREADS >>> (d_pos, d_vel, d_acc,  original_p_index, sorted_pos, sorted_vel, sorted_acc, num_parts);

	// hipDeviceSynchronize();
	copyparts_back<<< blks, NUM_THREADS >>> (parts, num_parts, d_pos, d_vel, d_acc);

}
